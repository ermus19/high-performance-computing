#include "hip/hip_runtime.h"
/*
 * dgemm_gpu_shared.cu
 *
 * compile with: make dgemm_gpu_shared_uvm
 *
 * Matrices are stored as array in row-major order: 
 * A[row][col] = A[row * N + col]
 *
 * Use shared memory to speed up the matrix multiplication. We can reuse
 * the memory if we load a block of the matrix and have a thread block 
 * calculate a sub matrix.
 */


#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

// Thread block size: BLOCK_SIZE * BLOCK_SIZE
#define BLOCK_SIZE 16

// Declaration of helper functions (see bottom of file for details)
void checkError (const char* action);
float getGflops (int, float);

/*
 *  Matrix multiplication kernel called by matrixMulOnDevice() 
 */
__global__ void dgemm_gpu_shared(double* a, double* b, double* c, int n){
    
    // TODO: Allocate shared memory for the two blocks aSub and bSub.
    //       Use two-dimensional matrices of size BLOCK_SIZE * BLOCK_SIZE 
    ... 
    
    // TODO: Calculate global thread index 
    int idxX = ... ;
    int idxY = ... ;
    
    // For the matrix multiplication, we need to multiply all the elements of 
    // the idxYth row of a with all the elements of the idXth column of b and 
    // sum up the results.
    double sum = 0;

    // TODO: Calculate global offset of upper left corner of thread block.
    int blockaY = ... ;
    int blockbX = ... ;

    for (int block = 0; block < gridDim.x; ++block){
        // Get the two sub matrices
        int blockaX = block * (BLOCK_SIZE);
        int blockbY = block * (BLOCK_SIZE);
        if (((blockaY + threadIdx.y) < n) && (blockaX + threadIdx.x) < n) {
          // TODO: Copy block into shared memory
	  ...
        } else {
            aSub[threadIdx.y][threadIdx.x] = 0;
        }

        if (((blockbY + threadIdx.y) < n) && (blockbX + threadIdx.x) < n) {
            bSub[threadIdx.y][threadIdx.x] = b[(blockbY + threadIdx.y) * n + blockbX + threadIdx.x];
        } else {
            bSub[threadIdx.y][threadIdx.x] = 0;
        }
	
	// TODO: Synchronize threads to make sure all threads are done copying
    
    
        if ((idxX < n) && (idxY < n))
        {
            for (int i=0; i < blockDim.x; ++i){ //assumes that we use square blocks
                sum += aSub[threadIdx.y][i] * bSub[i][threadIdx.x];
            }
        }

	// TODO: Synchronize threads to make sure all threads are done with the data

    }
    if ((idxX < n) && (idxY < n)){    
        c[idxY * n + idxX] = sum;
    }
}



/*
 *  Matrix multiplication host function called by main() 
 */

void matrixMulOnDevice(double* a, double* b, double* c, int n)
{
    int xGrid, yGrid;
    float time;

    // Define events for timing
    hipEvent_t start, stop;
  
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // First calculate grid size by dividing n by BLOCK_SIZE = 16
    xGrid = (n % BLOCK_SIZE == 0) ? (n / BLOCK_SIZE) : (n / BLOCK_SIZE + 1);
    yGrid = (n % BLOCK_SIZE == 0) ? (n / BLOCK_SIZE) : (n / BLOCK_SIZE + 1);
    dim3 gridDim(xGrid, yGrid);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

    printf("Grid: %d, %d; block:%d, %d\n", xGrid , yGrid , BLOCK_SIZE, BLOCK_SIZE);
    
    // Invoke kernel and measure execution time 
    hipEventRecord( start, 0 );
    

    // TODO: Call the kernel 
    ...
    hipDeviceSynchronize(); 

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    checkError("executing Kernel");
    
    // Get elapsed time for kernel execution
    hipEventElapsedTime( &time, start, stop );
    hipEventDestroy( start );
    hipEventDestroy( stop );

    printf ("\nKernel Execution Time: %f ms (dim C: %d * %d)", time, n, n);
    printf ("\nThis corresponds to: %4.4f GFLOPS", getGflops(n, time));
  
    

}

int main(int argc, char** argv)
{
    int n = 1024;
    double *a, *b, *c;
    int row, col;
    double absError, maxAbsError = 0.0, sumAbsError = 0.0;
    size_t size;
    if (argc > 1) {
       n = atoi(argv[1]);
    }

    // show banner
    printf ("\n\n     Matrix-Multiplication \n");
    printf (    "     ==========================================\n");
    printf (  "\n     Simple DGEMM implemantation on GPU");  

    // echo device data
    int idevice = 0;
    hipSetDevice(idevice);
    hipDeviceProp_t dprops;
    hipGetDeviceProperties( &dprops, idevice );
    printf ("\n     Device name = %s, with compute capability %d.%d \n", 
	    dprops.name, dprops.major, dprops.minor);
    printf (  "\n     Matrix size %d x %d", n, n);
  
    // TODO
    // Allocate memory for matrices (that can be accessed from host and device) 
    size = n * n * sizeof(double);
    ...
    checkError("hipMallocManaged: a");
    ...
    checkError("hipMallocManaged: b");  
    ...
    checkError("hipMallocManaged: c");
    
    // Init matrices A and B: A = E so result will be B
    #pragma omp parallel for private(row, col)
    for (row = 0; row < n; ++row){
      for (col = 0; col < n; col++){
	a[row * n + col] = (row == col) ? 1.0 : 0.0;
	b[row * n + col] = row * n + col;
      }
    }

    // do matrix multiplication on device
    matrixMulOnDevice(a, b, c, n);
     
    // Compare results
    for ( row = 0; row < n; ++row){
      for ( col = 0; col < n; ++col) {
	
	absError = fabs ( c[row * n + col] - b[row * n + col]);
	sumAbsError += absError;
	
	if (absError > maxAbsError)
	  maxAbsError = absError;
      }
    }
    // Free memory on host
    hipFree (a);
    hipFree (b);
    hipFree (c);
  
    printf ("\nmaxAbsError: %4.4f, sumAbsError: %4.4f", maxAbsError, sumAbsError);
    if (maxAbsError < 2.0e-5)
      printf ("\n\nProgram terminated SUCCESSFULLY.\n\n");

    return 0;
}

/*
 *  Some helper functions
 */

// get compute performance
float getGflops (int n, float time) {

	float gf = (2.0e-6 * n * n* n / time);

	return gf;
}

// Simple error checking function for CUDA actions

void checkError (const char* action) {
  
  hipError_t error;
  error = hipGetLastError(); 

  if (error != hipSuccess) {
    printf ("\nError while '%s': %s\nprogram terminated ...\n\n", action, hipGetErrorString(error));
    exit (EXIT_FAILURE);
  }
}
